#include "hip/hip_runtime.h"

#include "monte_carlo.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

monte_carlo::monte_carlo (std::vector < data_series > &historical_data, bool use_gpu):historical_data_ (historical_data), use_gpu_ (use_gpu)
{
	if (use_gpu_) {
		int num_stocks = historical_data_.size ();
		hipMallocManaged (&gpu_historical_data_, historical_data.size () * historical_data[0].size * sizeof (float));
		for (unsigned stock = 0; stock < historical_data_.size (); stock++) {
			memcpy (gpu_historical_data_ + stock * historical_data[0].size, historical_data_[stock].values, sizeof (float) * historical_data_[0].size);
		}
		hipMallocManaged (&gpu_portfolio_, num_stocks * sizeof (float));
	}
}

__global__ void run_simulation (int seed, const int num_rounds, int num_stocks, int num_days, float *historical_data, int start_day, int days_back, float *portfolio, float *expectancy_list)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init (seed,	/* the seed controls the sequence of random values that are produced */
		     idx,	/* the sequence number is only important with multiple cores */
		     0,		/* the offset is how much extra we advance in the sequence for each call, can be 0 */
		     &state);

	float p[512];

	memcpy (p, portfolio, num_stocks * sizeof (float));

	const int duration = 253 * 1 / 2;	// half a year @ 253 trading days per year

	int round = idx;
	int position, steps;
	steps = 0;
	for (int i = 0; i < duration; i++) {
		if ((steps % 30) == 0) {
			position = start_day + hiprand (&state) % (days_back - 1);
			steps = 0;
		}
		else {
			position++;
			steps++;
		}

		for (unsigned stock = 0; stock < num_stocks; stock++) {
			float factor = (historical_data[stock * num_days + position + 1] / historical_data[stock * num_days + position]);
			p[stock] *= factor;
		}
	}

	float expectancy = 0.;
	for (unsigned stock = 0; stock < num_stocks; stock++)
		expectancy += p[stock];

	expectancy_list[round] = expectancy * expectancy;
}

void monte_carlo::run_with_data (portfolio & p, std::vector < float >&expectancy_list, float &expectancy, float &standard_deviation, float &downside_deviation, int num_rounds, int days_back)
{
	int num_days = historical_data_[0].size;
	days_back = min (num_days, days_back);
	int start_day = num_days - days_back;

	if (use_gpu_) {
		int num_stocks = historical_data_.size ();

		float *gpu_expectancy_list;
		hipMallocManaged (&gpu_expectancy_list, num_rounds * sizeof (float));

		memcpy (gpu_portfolio_, p.proportions, num_stocks * sizeof (float));
		run_simulation <<< num_rounds / 256, 256 >>> (rand (), num_rounds, num_stocks, num_days, gpu_historical_data_, start_day, days_back, gpu_portfolio_, gpu_expectancy_list);

		hipDeviceSynchronize ();

		for (int i = 0; i < num_rounds; i++)
			expectancy_list.push_back (gpu_expectancy_list[i]);

		hipFree (gpu_expectancy_list);
	}
	else {
		const int duration = 253 * 1 / 2;	// half a year @ 253 trading days per year
		portfolio p2;

		for (int round = 0; round < num_rounds; round++) {
			p2 = p;

			int position, steps;
			steps = 0;
			for (int i = 0; i < duration; i++) {
				if ((steps % 30) == 0) {
					position = start_day + rand () % (days_back - 1);
					steps = 0;
				}
				else {
					position++;
					steps++;
				}

				for (unsigned stock = 0; stock < historical_data_.size (); stock++) {
					float factor = (historical_data_[stock].values[position + 1] / historical_data_[stock].values[position]);
					p2.proportions[stock] *= factor;
				}
			}

			float round_expectancy = 0.;
			for (unsigned stock = 0; stock < historical_data_.size (); stock++)
				round_expectancy += p2.proportions[stock];
			expectancy_list.push_back (round_expectancy);
		}
	}

	// calculate expectancy
	double expectancy_d = 0.f;
	for (int i = 0; i < num_rounds; i++) {
		expectancy_d += expectancy_list[i];
	}
	expectancy_d /= (double) num_rounds;
	expectancy = expectancy_d;

	// calculate standard deviation
	double standard_deviation_d = 0.f;
	for (int i = 0; i < num_rounds; i++)
		standard_deviation_d += (expectancy_list[i] - expectancy_d) * (expectancy_list[i] - expectancy_d);
	standard_deviation_d = sqrt (standard_deviation_d / (double) num_rounds);
	standard_deviation = standard_deviation_d;

	// calculate downside deviation
	double downside_deviation_d = 0.f;
	for (int i = 0; i < num_rounds; i++)
		if (expectancy_list[i] < expectancy)
			downside_deviation_d += (expectancy_list[i] - expectancy_d) * (expectancy_list[i] - expectancy_d);
	downside_deviation_d = sqrt (downside_deviation_d / (double) num_rounds);
	downside_deviation = downside_deviation_d;
}

void monte_carlo::run (portfolio & p, float &expectancy, float &standard_deviation, float &downside_deviation, int num_rounds, int days_back)
{
	std::vector < float >dummy_list;
	run_with_data (p, dummy_list, expectancy, standard_deviation, downside_deviation, num_rounds, days_back);
}
