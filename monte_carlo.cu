#include "hip/hip_runtime.h"

#include "monte_carlo.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

monte_carlo::monte_carlo(std::vector<data_series> & historical_data, bool use_gpu)
	: historical_data_ (historical_data), use_gpu_ (use_gpu)
{
	if (use_gpu_) {
		int num_stocks = historical_data_.size();
		hipMallocManaged(&gpu_historical_data_, historical_data.size() * historical_data[0].size * sizeof(float));
		for(unsigned stock = 0; stock < historical_data_.size(); stock++) {
			memcpy(gpu_historical_data_ + stock * historical_data[0].size, historical_data_[stock].values, sizeof(float) * historical_data_[0].size);
		}
		hipMallocManaged(&gpu_portfolio_, num_stocks * sizeof(float));
	}
}

__global__ void run_simulation(int seed, const int num_rounds, int num_stocks, int num_days, float *historical_data, float *portfolio, float *expectancy_list)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
			idx, /* the sequence number is only important with multiple cores */
			0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&state);

	float p[512];

	memcpy(p, portfolio, num_stocks * sizeof(float));

	const int duration = 253 * 1; // 1 years @ 253 trading days per year

	int round = idx;
	for(int i = 0; i < duration; i++) {
		int position = hiprand(&state) % (num_days - 1);

		for(unsigned stock = 0; stock < num_stocks; stock++) {
			float factor = (historical_data[stock * num_days + position + 1] / historical_data[stock * num_days + position]);
			p[stock] *= factor;
		}
	}

	float expectancy = 0.;
	for(unsigned stock = 0; stock < num_stocks; stock++)
		expectancy += p[stock];

	expectancy_list[round] = expectancy;
}

void monte_carlo::run(portfolio &p, float &expectancy, float &standard_deviation, int num_rounds)
{
	if (use_gpu_) {
		int num_stocks = historical_data_.size();
		int num_days = historical_data_[0].size;

		float *expectancy_list;
		hipMallocManaged(&expectancy_list, num_rounds * sizeof(float)); 

		memcpy(gpu_portfolio_, p.proportions, num_stocks * sizeof(float));
		run_simulation<<<num_rounds / 256, 256>>>(rand(), num_rounds, num_stocks, num_days, gpu_historical_data_, gpu_portfolio_, expectancy_list);

		hipDeviceSynchronize();

		// calculate expectancy
		expectancy = 0.f;
		for(int i = 0; i < num_rounds; i++) {
			expectancy += expectancy_list[i];
		}
		expectancy /= (float) num_rounds;

		// calculate standard deviation
		standard_deviation = 0.f;
		for(int i = 0; i < num_rounds; i++)
			standard_deviation += (expectancy_list[i] - expectancy) * (expectancy_list[i] - expectancy);
		standard_deviation = sqrtf(standard_deviation / (float) num_rounds);
	} else {
		const int duration = 253 * 1; // 1 years @ 253 trading days per year
		std::vector<float> expectancy_list;
		portfolio p2;

		for(int round = 0; round < num_rounds; round++) {
	 		p2 = p;

			for(int i = 0; i < duration; i++) {
				int position = rand() % (historical_data_[0].size - 1);
	
				for(unsigned stock = 0; stock < historical_data_.size(); stock++) {
					float factor = (historical_data_[stock].values[position + 1] / historical_data_[stock].values[position]);
					p2.proportions[stock] *= factor;
				}
			}

			float round_expectancy = 0.;
			for(unsigned stock = 0; stock < historical_data_.size(); stock++)
				round_expectancy += p2.proportions[stock];
			expectancy_list.push_back(round_expectancy);
		}

		// calculate expectancy
		expectancy = 0.f;
		for(unsigned i = 0; i < expectancy_list.size(); i++)
			expectancy += expectancy_list[i];
		expectancy /= (float) expectancy_list.size();

		// calculate standard deviation
		standard_deviation = 0.f;
		for(unsigned i = 0; i < expectancy_list.size(); i++)
			standard_deviation += (expectancy_list[i] - expectancy) * (expectancy_list[i] - expectancy);
		standard_deviation = sqrtf(standard_deviation / (float) expectancy_list.size());
	}
}

