#include "hip/hip_runtime.h"

#include "monte_carlo.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

monte_carlo::monte_carlo (std::vector < data_series > &historical_data, bool use_gpu):historical_data_ (historical_data), use_gpu_ (use_gpu)
{
	if (use_gpu_) {
		int num_stocks = historical_data_.size ();
		hipMallocManaged (&gpu_historical_data_, historical_data.size () * historical_data[0].size * sizeof (float));
		for (unsigned stock = 0; stock < historical_data_.size (); stock++) {
			memcpy (gpu_historical_data_ + stock * historical_data[0].size, historical_data_[stock].values, sizeof (float) * historical_data_[0].size);
		}
		hipMallocManaged (&gpu_portfolio_, num_stocks * sizeof (float));
	}
}

__global__ void run_simulation (int seed, const int num_rounds, int num_stocks, int num_days, float *historical_data, int start_day, int days_back, float *portfolio, float *expectancy_list)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t state;

	hiprand_init (seed, idx, 0, &state);

	float p[GPU_SIMULATION_MAX_STOCKS];

	memcpy (p, portfolio, num_stocks * sizeof (float));

	const int duration = MONTE_CARLO_SIMULATION_DURATION;

	int round = idx;
	int position;
	for (int i = 0; i < duration; i++) {
		position = (start_day + hiprand (&state) % (days_back - 1)) % num_days;

		for (unsigned stock = 0; stock < num_stocks; stock++) {
			float factor = (historical_data[stock * num_days + position + 1] / historical_data[stock * num_days + position]);
			p[stock] *= factor;
		}
	}

	float expectancy = 0.;
	for (unsigned stock = 0; stock < num_stocks; stock++)
		expectancy += p[stock];

	// We simulated half a year, but expectanty is easier to read as returns per year
	expectancy_list[round] = expectancy * expectancy;
}

void monte_carlo::run_with_data (portfolio & p, std::vector < float >&expectancy_list, float &expectancy, float &standard_deviation, float &downside_deviation, int num_rounds, int days_back)
{
	int num_days = historical_data_[0].size;
	days_back = min (num_days, days_back);
	int start_day = num_days - days_back;

	if (use_gpu_) {
		int num_stocks = historical_data_.size ();
		assert (num_stocks < GPU_SIMULATION_MAX_STOCKS);

		float *gpu_expectancy_list;
		hipMallocManaged (&gpu_expectancy_list, num_rounds * sizeof (float));

		memcpy (gpu_portfolio_, p.proportions, num_stocks * sizeof (float));
		run_simulation <<< num_rounds / 256, 256 >>> (rand (), num_rounds, num_stocks, num_days, gpu_historical_data_, start_day, days_back, gpu_portfolio_, gpu_expectancy_list);

		hipDeviceSynchronize ();

		for (int i = 0; i < num_rounds; i++)
			expectancy_list.push_back (gpu_expectancy_list[i]);

		hipFree (gpu_expectancy_list);
	}
	else {
		const int duration = MONTE_CARLO_SIMULATION_DURATION;
		portfolio p2;

		for (int round = 0; round < num_rounds; round++) {
			p2 = p;

			int position;
			for (int i = 0; i < duration; i++) {
				position = (start_day + rand () % (days_back - 1)) % num_days;

				for (unsigned stock = 0; stock < historical_data_.size (); stock++) {
					float factor = (historical_data_[stock].values[position + 1] / historical_data_[stock].values[position]);
					p2.proportions[stock] *= factor;
				}
			}

			float round_expectancy = 0.;
			for (unsigned stock = 0; stock < historical_data_.size (); stock++)
				round_expectancy += p2.proportions[stock];

			// We simulated half a year, but expectanty is easier to read as returns per year
			expectancy_list.push_back (round_expectancy * round_expectancy);
		}
	}

	// calculate expectancy
	double expectancy_d = 0.f;
	for (int i = 0; i < num_rounds; i++)
		expectancy_d += expectancy_list[i];
	expectancy_d /= (double) num_rounds;
	expectancy = expectancy_d;

	// calculate standard deviation
	double standard_deviation_d = 0.f;
	for (int i = 0; i < num_rounds; i++)
		standard_deviation_d += (expectancy_list[i] - expectancy_d) * (expectancy_list[i] - expectancy_d);
	standard_deviation_d = sqrt (standard_deviation_d / (double) num_rounds);
	standard_deviation = standard_deviation_d;

	// calculate downside deviation
	double downside_deviation_d = 0.f;
	for (int i = 0; i < num_rounds; i++)
		if (expectancy_list[i] < expectancy)
			downside_deviation_d += (expectancy_list[i] - expectancy_d) * (expectancy_list[i] - expectancy_d);
	downside_deviation_d = sqrt (downside_deviation_d / (double) num_rounds);
	downside_deviation = downside_deviation_d;
}

void monte_carlo::run (portfolio & p, float &expectancy, float &standard_deviation, float &downside_deviation, int num_rounds, int days_back)
{
	std::vector < float >dummy_list;
	run_with_data (p, dummy_list, expectancy, standard_deviation, downside_deviation, num_rounds, days_back);
}
